
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#define CUDACHECK(err) { cuda_check((err), __FILE__, __LINE__); }

inline void cuda_check(hipError_t error_code, const char *file, int line)
{
    if (error_code != hipSuccess)
    {
        fprintf(stderr, "CUDA Error %d: %s. In '%s' on line %d\n", error_code, hipGetErrorString(error_code), file, line);
        exit(error_code);
    }
}

#define CUDAMEASURE(command) do { hipEvent_t b,e; \
    CUDACHECK(hipEventCreate(&b)); CUDACHECK(hipEventCreate(&e)); \
    CUDACHECK(hipEventRecord(b)); command ; CUDACHECK(hipEventRecord(e)); \
    CUDACHECK(hipEventSynchronize(e)); \
    float time; CUDACHECK(hipEventElapsedTime(&time, b, e)); \
    printf("Execution time: %f ms\n", time); \
    CUDACHECK(hipEventDestroy(b)); CUDACHECK(hipEventDestroy(e)); } while(false)


// Draws a glider on the grid skipping col_offst number of cols from left and ...
void create_glider(int col_offst, int row_offst, int grid_width, int grid_height, int *grid) {
    int glider_coords[3][3] = {{0, 0, 1}, {1, 0, 1}, {0, 1, 1}};
    for(int x_pos = 0; x_pos < 3; x_pos++) {
        for(int y_pos = 0; y_pos < 3; y_pos++) {
            if(col_offst<=grid_width-3 && row_offst<=grid_height-3) {
                grid[(x_pos+row_offst)*grid_width + (y_pos+col_offst)] = glider_coords[x_pos][y_pos];
            } else {
                std::cout<<"glider out of grid"<<std::endl;
            }
        }
    }
}

void print_grid(int grid_width, int grid_height, int *grid) {
    for (int i = 0; i< grid_width; i++) {
        for (int j = 0; j< grid_height; j++) {
            if (grid[i*grid_width + j] == 0) {
                std::cout<<" - ";
            } else {
                std::cout<<" X ";
            }
        }
        std::cout<<std::endl;
    }
}

__global__ void run_life_one_step(const int* world_matrix, int grid_width,
                                  int grid_height, int* result_world_matrix) {
    int worldSize = grid_width * grid_height;

    for (int cellId = (blockIdx.x * blockDim.x) + threadIdx.x; cellId < worldSize; cellId += blockDim.x * gridDim.x) {
        int x_pos = cellId % grid_width;
        int y_pos = cellId - x_pos;

        int upper_cell_pos = (y_pos + worldSize - grid_width) % worldSize;
        int lower_cell_pos = (y_pos + grid_width) % worldSize;
        int left_cell_pos = (x_pos + grid_width - 1) % grid_width;
        int right_cell_pos = (x_pos + 1) % grid_width;

        int neighbour_sum = world_matrix[left_cell_pos + upper_cell_pos] + world_matrix[x_pos + upper_cell_pos]
                          + world_matrix[right_cell_pos + upper_cell_pos] + world_matrix[left_cell_pos + y_pos] + world_matrix[right_cell_pos + y_pos]
                          + world_matrix[left_cell_pos + lower_cell_pos] + world_matrix[x_pos + lower_cell_pos] + world_matrix[right_cell_pos + lower_cell_pos];

        result_world_matrix[x_pos + y_pos] = neighbour_sum == 3 || (neighbour_sum == 2 && world_matrix[x_pos + y_pos]) ? 1 : 0;
    }
}

void run_life(int*& d_world_matrix, int*& d_world_matrixBuffer, size_t grid_width,
              size_t grid_height, size_t iterationsCount, short threadsCount) {

    int reqBlocksCount = (grid_width * grid_height) / threadsCount;
    int blocksCount = (int) std::min(65535, reqBlocksCount);

    for (int i = 0; i < iterationsCount; ++i) {
        run_life_one_step<<<blocksCount, threadsCount>>>(d_world_matrix, grid_width, grid_height, d_world_matrixBuffer);
        std::swap(d_world_matrix, d_world_matrixBuffer);
    }
}

int main(int argc, char **argv) {
    int grid_width = 30;
    int grid_height = 30;
    int generations = 1000;
    int tpb = 32;

    for (int i = 0; i < argc; ++i) {
        if (!strcmp("-n", argv[i])) {
            grid_width = atoi(argv[i + 1]);
        } else if (!strcmp("-m", argv[i])) {
            grid_height = atoi(argv[i + 1]);
        } else if (!strcmp("-max", argv[i])) {
            generations = atoi(argv[i + 1]);
        } else if (!strcmp("-tpb", argv[i])) {
            tpb = atoi(argv[i + 1]);
        }
    }
    size_t size = sizeof(float)*grid_height*grid_width;

    int* life_data_host;
    int* life_data_buffer_host;

    CUDACHECK(hipHostMalloc(&life_data_host, size, hipHostMallocDefault));
    CUDACHECK(hipHostMalloc(&life_data_buffer_host, size, hipHostMallocDefault));

    create_glider(3, 4, grid_width, grid_height, life_data_host);
    create_glider(0, 12, grid_width, grid_height, life_data_host);
    print_grid(grid_width, grid_height, life_data_host);

    int* life_data_device;
    int* life_data_buffer_device;

    CUDACHECK(hipMalloc(&life_data_device, size));
    CUDACHECK(hipMalloc(&life_data_buffer_device, size));


    CUDACHECK(hipMemcpy(life_data_device, life_data_host, size, hipMemcpyHostToDevice));
    CUDACHECK(hipMemcpy(life_data_buffer_device, life_data_buffer_host, size, hipMemcpyHostToDevice));

    hipEventCreaten_life(life_data_device, life_data_buffer_device, grid_width, grid_height, generations, tpb)));
    CUDACHECK(hipMemcpy(life_data_host, life_data_device, size, hipMemcpyDeviceToHost));

    CUDACHECK(hipDeviceSynchronize());

    print_grid(grid_width, grid_height, life_data_host);

    return 0;
}

